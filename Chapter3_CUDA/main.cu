#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "../Utils/bmpimage.h"
#include "../Utils/vector3.h"

#include <stdio.h>
#include <iostream>

//__constant__ static const float kPI = 3.1415927f;

//__device__ inline float DegToRad(float Deg)
//{
//	return (Deg * kPI / 180.0f);
//}


//Ray tracing data structures

//Simple struct used to collect post hit data (i.e. hit position, normal and t)
struct HitData
{
	/** Ctor */
	__device__ HitData() : mHitPos(0.f,0.f,0.f), mNormal(0.f,1.f,0.f) { }
	
	Vector3 mHitPos;
	
	Vector3 mNormal;
	
	float t = 0.0f;
};

class Camera
{
public:

	__device__ Camera(const Vector3& InEye = Vector3(0.f, 0.f, 0.f), const Vector3& InLookAt = Vector3(0.f, 0.f, 50.f), const Vector3& InUp = Vector3(0.f, 1.f, 0.f), float InFov = 60.f, float InAspectRatio = 1.f) : mEye(InEye), mLookAt(InLookAt)
	{

		const Vector3& Fwd = InLookAt - InEye;
		mW = Fwd.norm();
		mU = mW.cross(InUp);
		mV = mU.cross(mW);

		mScaleY = tanf(DegToRad(InFov)*0.5f);
		mScaleX = mScaleY * InAspectRatio;
	}

	

	~Camera() = default;

	//We calculate the world space ray given the position of the pixel in image space and 
	//the image plane width and height.
	__device__ Vector3 GetWorldSpaceRayDir(float InPx, float InPy, float InWidth, float InHeight)
	{
		float Alpha = ((InPx / InWidth)*2.0f - 1.0f)*mScaleX;
		float Beta = ((1.0f - (InPy / InHeight))*2.0f - 1.0f)*mScaleY;

		Vector3 WSRayDir = mU * Alpha + mV * Beta + mW;

		return WSRayDir;
	}

	__device__ Vector3 GetCameraEye() const { return mEye; }

	//we could add more accessor (getter/setter) if necessary

private:

	//Convenient member variables used to cache the scale along the x and y axis of the
	//camera space

	float mScaleY = 1.0f;

	float mScaleX = 1.0f;

	/**The camera position */
	Vector3 mEye;
	/**The camera forward vector  */
	Vector3 mW;
	/**The camera side vector*/
	Vector3 mU;
	/**The camera up vector */
	Vector3 mV;
	/**The camera look at */
	Vector3 mLookAt;

};


//Simple ray class 
class Ray
{
public:
	
	/** Ctor */
	__device__ Ray(const Vector3& InOrigin = Vector3(0, 0, 0), const Vector3& InDirection = Vector3(0, 0, 1)) : mOrigin(InOrigin), mDirection(InDirection) {}

	/** Copy Ctor */
	__device__ Ray(const Ray& InRay) : mOrigin(InRay.mOrigin), mDirection(InRay.mDirection) { }

	//Method used to compute position at parameter t
	__device__ Vector3 PositionAtT(float t) const
	{
		return mOrigin + mDirection * t;
	}

	Vector3 mOrigin;

	Vector3 mDirection;

	float mTmin;

	float mTmax;

};


//Simple sphere class
class Sphere
{
private:

	/** The center of the sphere */
	Vector3 mCenter;

	/** The radius of the sphere */
	float mRadius;

public:

	/** Ctor */
	__device__ Sphere(const Vector3& InCenter = Vector3(0, 0, 0), float InRadius = 1) : mCenter(InCenter), mRadius(InRadius) {  }

	/** Copy Ctor */
	__device__ Sphere(const Sphere& InSphere) : mCenter(InSphere.mCenter), mRadius(InSphere.mRadius) {  }

	//Compute the ray-sphere intersection using analitic solution
	__device__ bool Intersect(const Ray& InRay, float InTMin, float InTMax, HitData& OutHitData)
	{
		const Vector3& oc = (InRay.mOrigin - mCenter);
		float a = InRay.mDirection.dot(InRay.mDirection);
		float b = oc.dot(InRay.mDirection);
		float c = oc.dot(oc) - mRadius * mRadius;
		float Disc = b * b - a * c;
		float SqrtDisc = sqrt(Disc);
		if (Disc > 0)
		{
			float temp = (-b - SqrtDisc) / a;
			if (temp < InTMax && temp > InTMin)
			{
				OutHitData.t = temp;
				OutHitData.mHitPos = InRay.PositionAtT(temp);
				OutHitData.mNormal = (OutHitData.mHitPos - mCenter) / mRadius;
				return true;
			}
			temp = (-b + SqrtDisc) / a;
			if (temp < InTMax && temp > InTMin)
			{
				OutHitData.t = temp;
				OutHitData.mHitPos = InRay.PositionAtT(temp);
				OutHitData.mNormal = (OutHitData.mHitPos - mCenter) / mRadius;
				return true;
			}

		}
		return false;
	}
};



//Useful macro to check cuda error code returned from cuda functions
#define CHECK_CUDA_ERRORS(val) Check( (val), #val, __FILE__, __LINE__ )
static void Check(hipError_t result, char const *const func, const char *const file, int const line)
{
	if (result)
	{
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' \n";
		hipDeviceReset();
		exit(99);
	}
}

//the keyword __global__ instructs the CUDA compiler that this function is the entry point of our kernel
__global__ void RenderScene(const int N, float* ColorBuffer)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	//checks whether we are inside the color buffer bounds.
	//If not, just return
	if (x >= N || y >= N)
	{
		return;
	}

	//Create a simple sphere 10 units away from the world origin
	Sphere sphere(Vector3(0.0f,0.0f,1.0f),1.0f);

	//Prepare two color
	Vector3 Black(0.0f, 0.0f, 0.0f);   //Black background if we miss a primitive
	Vector3 Green(0.0f, 1.0f, 0.0f);  //Red color if we hit a primitive (in our case a sphere, but can be any type of primitive)

	//Create a camera
	Camera camera(Vector3(0.0f,0.0f,-5.0f));

	//Cast a ray in world space from the camera

	//Compute the world space ray direction
	auto WSDir = camera.GetWorldSpaceRayDir(x,y,N,N);

	//Construct a ray in world space that originates from the camera
	Ray WSRay(camera.GetCameraEye(), WSDir);

	//Compute intersection and set a color
	HitData OutHitData;
	Vector3 ColorResult = sphere.Intersect(WSRay,0.001f,FLT_MAX,OutHitData) ? Green : Black;


	//We access the linear ColorBuffer storing each color component separately (we could have a float3    color buffer for a more compact/cleaner solution)
	int offset = (x + y * N) * 3;

	//Store the results of your computations
	ColorBuffer[offset] = ColorResult.X();
	ColorBuffer[offset + 1] = ColorResult.Y();
	ColorBuffer[offset + 2] = ColorResult.Z();
}

int main()
{
	//Color Buffer resolution
	int ScreenWidth = 512;
	int ScreenHeight = 512;

	float* ColorBuffer = nullptr;

	//Here we prepare our computation domain (i.e. thread blocks and threads in a block)

	//Number of threads in a block (experiment with this sizes!). 
	//Suggenstion: make them a multiple of a warp (a warp is 32 threads wide on NVIDIA and 64 threads on AMD)
	int ThreadBlockSizeX = 8;
	int ThreadBlockSizeY = 8;

	//Number of thread blocks
	int NumOfBlockX = ScreenWidth / ThreadBlockSizeX + 1;
	int NumOfBlockY = ScreenHeight / ThreadBlockSizeY + 1;

	//Let's define the compute dimention domain
	dim3 ThreadBlocks(NumOfBlockX, NumOfBlockY);
	dim3 ThreadsInABlock(ThreadBlockSizeX, ThreadBlockSizeY);

	//Color buffer size in bytes
	const size_t kColorBufferSize = sizeof(float) * 3 * ScreenWidth*ScreenHeight;

	//We allocate our color buffer in Unified Memory such that it'll be easy for us to access it on the host as well as on the device
	CHECK_CUDA_ERRORS(hipMallocManaged(&ColorBuffer, kColorBufferSize));

	//Launch the kernel that will render the scene
	RenderScene << <ThreadBlocks, ThreadsInABlock >> > (ScreenWidth, ColorBuffer);

	//Wait for the GPU to finish before to access results on the host 
	CHECK_CUDA_ERRORS(hipGetLastError());
	CHECK_CUDA_ERRORS(hipDeviceSynchronize());


	//Save results stored in ColorBuffer to file (could be a *.ppx or a *.bmp)	

	//We are ready to use the results produced on the GPU
	//Dump Results on a file 
	const int dpi = 72;
	Ray_BMP_Manager::Save("Chapter3_CudaResult.bmp", ScreenWidth, ScreenHeight, dpi, (float*)ColorBuffer);

	//Done! Free up cuda allocated memory
	CHECK_CUDA_ERRORS(hipFree(ColorBuffer));

	return 0;
}