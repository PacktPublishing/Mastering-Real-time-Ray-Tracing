#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "bmpimage.h"

#include <stdio.h>
#include <iostream>


//Useful macro to check cuda error code returned from cuda functions
#define CHECK_CUDA_ERRORS(val) Check( (val), #val, __FILE__, __LINE__ )
static void Check(hipError_t result, char const *const func, const char *const file, int const line)
{
	if (result)
	{
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' \n";
		hipDeviceReset();
		exit(99);
	}
}

//Here we overload few useful math functions/operators in order to use them in our filter kernel
__device__ float3 operator+(const float3 &a, const float3 &b) {

	return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);

}

__device__ float3 operator*(const float3 &a, const float3 &b) {

	return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);

}

__device__ float3 operator/(const float3 &a, const float3 &b) {

	return make_float3(a.x / b.x, a.y / b.y, a.z / b.z);

}

__device__ float3 operator/(const float3 &a, const float b) {

	return make_float3(a.x / b, a.y / b, a.z / b);

}

__device__ float3 operator*(const float3 &a, const float b) {

	return make_float3(a.x * b, a.y * b, a.z * b);

}

__device__ int Max(int a, int b)
{
	return a > b ? a : b;
}


__device__ int Min(int a, int b)
{
	return a < b ? a : b;
}

__constant__ const int kFilterRadius = 25;
__constant__ const float FilterWeights[25] = { 0.f,	0.f,	0.000001f,	0.00001f,	0.000078f,	0.000489f,	0.002403f,	0.009245f,	0.027835f,	0.065591f,	0.120978f,	0.174666f,	0.197413f,	0.174666f,	0.120978f,	0.065591f,	0.027835f,	0.009245f,	0.002403f,	0.000489f,	0.000078f,	0.00001f,	0.000001f,	0.f,	0.f };

//the keyword __global__ instructs the CUDA compiler that this function is the entry point of our kernel
__global__ void FilterImageKernel(float* ColorBuffer
	                            , const int Width	
	                            , const int Height
	                            , float* FilteredColorBuffer)
{
	//shared memory 
	__shared__ float3 CachedColors[512];
	
	//Compute global x and t coords
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	//checks whether we are inside the color buffer bounds.
    //If not, just return
	if (x >= Width || y >= Height)
	{
		return;
	}

	const int ColorBufferOffset = (x + y * Width) * 3;

	//Start memory transactions here (copy color from global memory to shared fast memory)       
	CachedColors[threadIdx.x].x = ColorBuffer[ColorBufferOffset   ];
	CachedColors[threadIdx.x].y = ColorBuffer[ColorBufferOffset +1];
	CachedColors[threadIdx.x].z = ColorBuffer[ColorBufferOffset +2];

	
	//wait for all the threads in the block to finish their memory transactions before accessing any value store in CachedColors
	__syncthreads();


	//Add filter code here
	int OffsetThreadId = threadIdx.x - kFilterRadius / 2;
	float3 Result = make_float3(0.0f, 0.0f, 0.0f);
	for (int x = 0; x < kFilterRadius; ++x)
	{		
		Result = Result + (CachedColors[Min(Max(0,OffsetThreadId + x),Width-1)] * FilterWeights[x]);
	}
	//write back the filter result in global memory rotating 90� the image (this is a trick to always have coalesced access pattern on global memory read)
	const int RotatedOffset = (y + x * Width) * 3;	
	FilteredColorBuffer[RotatedOffset    ] = Result.x;
	FilteredColorBuffer[RotatedOffset + 1] = Result.y;
	FilteredColorBuffer[RotatedOffset + 2] = Result.z;
}


int main()
{

	float* ColorBuffer = nullptr;
	float* IntermediateResults = nullptr;

	//Here we prepare our computation domain (i.e. thread blocks and threads in a block)

	//Number of threads in a block (experiment with this sizes!). 
	//Suggenstion: make them a multiple of a warp (a warp is 32 threads wide on NVIDIA and 64 threads on AMD)
	int ThreadBlockSizeX = 512;
	int ThreadBlockSizeY = 1;

	//Image Buffer default resolution
	int ImageWidth = 256;
	int ImageHeight = 256;


	//Load an bmp image
	float* ImageData = Ray_BMPSaver::Load("flower.bmp", ImageWidth, ImageHeight);

	//Number of thread blocks
	int NumOfBlockX = ImageWidth / ThreadBlockSizeX;
	int NumOfBlockY = ImageHeight / ThreadBlockSizeY;

	//Let's define the compute dimention domain
	dim3 ThreadBlocks(NumOfBlockX, NumOfBlockY);
	dim3 ThreadsInABlock(ThreadBlockSizeX, ThreadBlockSizeY);	

	//Color buffer size in bytes
	const size_t kColorBufferSize = sizeof(float) * 3 * ImageWidth*ImageHeight;


	//We allocate our color buffer in Unified Memory such that it'll be easy for us to access it on the host as well as on the device
	CHECK_CUDA_ERRORS(hipMallocManaged(&ColorBuffer, kColorBufferSize));
	CHECK_CUDA_ERRORS(hipMallocManaged(&IntermediateResults, kColorBufferSize));

	//copy image data from host memory to device memory for processing
	memcpy(ColorBuffer, ImageData, kColorBufferSize);

	//Perform horizontal blur pass
	FilterImageKernel << <ThreadBlocks, ThreadsInABlock >> > (ColorBuffer, ImageWidth, ImageHeight,IntermediateResults);

	//Wait for the GPU to finish before to access results of the previous pass
	CHECK_CUDA_ERRORS(hipGetLastError());
	CHECK_CUDA_ERRORS(hipDeviceSynchronize());

	//Perform vertical blur pass
	FilterImageKernel << <ThreadBlocks, ThreadsInABlock >> > (IntermediateResults, ImageWidth, ImageHeight, ColorBuffer);

	//Wait for the GPU to finish before to access results of the final pass
	CHECK_CUDA_ERRORS(hipGetLastError());
	CHECK_CUDA_ERRORS(hipDeviceSynchronize());


	//Save results stored in ColorBuffer to file (could be a *.ppx or a *.bmp)	

	//We are ready to use the results produced on the GPU
	//Dump Results on a file 
	const int dpi = 72;
	Ray_BMPSaver::Save("Chapter2_CudaResult.bmp", ImageWidth, ImageHeight, dpi, (float*)ColorBuffer);

	//Done! Free up cuda allocated memory
	CHECK_CUDA_ERRORS(hipFree(IntermediateResults));
	CHECK_CUDA_ERRORS(hipFree(ColorBuffer));

	return 0;
}

