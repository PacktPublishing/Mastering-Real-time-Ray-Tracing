#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "bmpimage.h"

#include <stdio.h>
#include <iostream>


//Useful macro to check cuda error code returned from cuda functions
#define CHECK_CUDA_ERRORS(val) Check( (val), #val, __FILE__, __LINE__ )
static void Check(hipError_t result, char const *const func, const char *const file, int const line)
{
	if (result)
	{
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' \n";
		hipDeviceReset();
		exit(99);
	}
}

__device__ float3 operator+(const float3 &a, const float3 &b) {

	return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);

}

__device__ float3 operator*(const float3 &a, const float3 &b) {

	return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);

}

__device__ float3 operator/(const float3 &a, const float3 &b) {

	return make_float3(a.x / b.x, a.y / b.y, a.z / b.z);

}

__device__ float3 operator/(const float3 &a, const float b) {

	return make_float3(a.x / b, a.y / b, a.z / b);

}

__device__ float3 operator*(const float3 &a, const float b) {

	return make_float3(a.x * b, a.y * b, a.z * b);

}

__device__ int Max(int a, int b)
{
	return a > b ? a : b;
}


__device__ int Min(int a, int b)
{
	return a < b ? a : b;
}

__device__ __constant__ const int kFilterRadius = 25;
__device__ __constant__ const float FilterWeights[kFilterRadius] = { 0,	0,	0.000001,	0.00001,	0.000078,	0.000489,	0.002403,	0.009245,	0.027835,	0.065591f,	0.120978,	0.174666,	0.197413,	0.174666,	0.120978,	0.065591,	0.027835,	0.009245,	0.002403,	0.000489,	0.000078,	0.00001,	0.000001,	0,	0 };

//the keyword __global__ instructs the CUDA compiler that this function is the entry point of our kernel
__global__ void FilterImageKernel(float* ColorBuffer
	                            , const int Width	                                                      
	                            , float* FilteredColorBuffer)
{
	//shared memory 
	__shared__ float3 CachedColors[512];


	//Compute global x and t coords
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	//checks whether we are inside the color buffer bounds.
    //If not, just return
	if (x >= Width || y >= Width)
	{
		return;
	}

	const int ColorBufferOffset = (x + y * Width) * 3;

	//Start memory transactions here (copy color from global memory to shared fast memory)       
	CachedColors[threadIdx.x].x = ColorBuffer[ColorBufferOffset   ];
	CachedColors[threadIdx.x].y = ColorBuffer[ColorBufferOffset +1];
	CachedColors[threadIdx.x].z = ColorBuffer[ColorBufferOffset +2];

	
	//wait for all the threads in the block to finish their memory transactions before accessing any value store in CachedColors
	__syncthreads();

	//FilteredColorBuffer[ColorBufferOffset] = CachedColors[threadIdx.x].x;
	//FilteredColorBuffer[ColorBufferOffset + 1] = CachedColors[threadIdx.x].y;
	//FilteredColorBuffer[ColorBufferOffset + 2] = CachedColors[threadIdx.x].z;

	//return;

	//Add filter code here
	int OffsetThreadId = threadIdx.x - kFilterRadius / 2;
	float3 Result = make_float3(0.0f, 0.0f, 0.0f);
	for (int x = 0; x < kFilterRadius; ++x)
	{		
		Result = Result + (CachedColors[Min(Max(0,OffsetThreadId + x),Width-1)] * FilterWeights[x]);
	}
	//write back the filter result in global memory rotating 90� the image (this is a trick to always have coalesced access pattern on global memory read)
	const int RotatedOffset = (y + x * Width) * 3;	
	FilteredColorBuffer[RotatedOffset    ] = Result.x;
	FilteredColorBuffer[RotatedOffset + 1] = Result.y;
	FilteredColorBuffer[RotatedOffset + 2] = Result.z;
}


int main()
{

	float* ColorBuffer = nullptr;
	float* IntermediateResults = nullptr;

	//Here we prepare our computation domain (i.e. thread blocks and threads in a block)

	//Number of threads in a block (experiment with this sizes!). 
	//Suggenstion: make them a multiple of a warp (a warp is 32 threads wide on NVIDIA and 64 threads on AMD)
	int ThreadBlockSizeX = 512;
	int ThreadBlockSizeY = 1;

	//Image Buffer default resolution
	int ImageWidth = 256;
	int ImageHeight = 256;


	//Load an bmp image
	float* ImageData = Ray_BMPSaver::Load("flower.bmp", ImageWidth, ImageHeight);

	//Number of thread blocks
	int NumOfBlockX = ImageWidth / ThreadBlockSizeX;
	int NumOfBlockY = ImageHeight / ThreadBlockSizeY;

	//Let's define the compute dimention domain
	dim3 ThreadBlocks(NumOfBlockX, NumOfBlockY);
	dim3 ThreadsInABlock(ThreadBlockSizeX, ThreadBlockSizeY);


	//Color buffer size in bytes
	const size_t kColorBufferSize = sizeof(float) * 3 * ImageWidth*ImageHeight;


	//We allocate our color buffer in Unified Memory such that it'll be easy for us to access it on the host as well as on the device
	CHECK_CUDA_ERRORS(hipMallocManaged(&ColorBuffer, kColorBufferSize));
	CHECK_CUDA_ERRORS(hipMallocManaged(&IntermediateResults, kColorBufferSize));

	//copy image data from host memory to device memory for processing
	memcpy(ColorBuffer, ImageData, kColorBufferSize);

	//Perform horizontal blur pass
	FilterImageKernel << <ThreadBlocks, ThreadsInABlock >> > (ColorBuffer, ImageWidth,IntermediateResults);

	//Wait for the GPU to finish before to access results of the previous pass
	CHECK_CUDA_ERRORS(hipGetLastError());
	CHECK_CUDA_ERRORS(hipDeviceSynchronize());

	//Perform vertical blur pass
	FilterImageKernel << <ThreadBlocks, ThreadsInABlock >> > (IntermediateResults, ImageWidth, ColorBuffer);

	//Wait for the GPU to finish before to access results of the final pass
	CHECK_CUDA_ERRORS(hipGetLastError());
	CHECK_CUDA_ERRORS(hipDeviceSynchronize());


	//Save results stored in ColorBuffer to file (could be a *.ppx or a *.bmp)	

	//We are ready to use the results produced on the GPU
	//Dump Results on a file 
	const int dpi = 72;
	Ray_BMPSaver::Save("Chapter2_CudaResult.bmp", ImageWidth, ImageHeight, dpi, (float*)ColorBuffer);

	//Done! Free up cuda allocated memory
	CHECK_CUDA_ERRORS(hipFree(IntermediateResults));
	CHECK_CUDA_ERRORS(hipFree(ColorBuffer));

	return 0;
}

